/*
    GPU上矩阵乘法
*/

//主机端函数
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <chrono>
using namespace std;
using namespace chrono;
//CUDA RunTime API
#include <hip/hip_runtime.h>
//单个 block 大小
#define THREAD_NUM 256
//矩阵大小
#define ROW_A 100
#define COL_A 100
#define ROW_B 100
#define COL_B 100
#define RANGE 10
//block个数
int blocks_num = (ROW_A * COL_B + THREAD_NUM - 1) / THREAD_NUM + 1;

//生成矩阵
void generateMatrix(float *A, float *B)
{
    srand((int)time(NULL));
    for (int i = 0; i < ROW_A; i++)
    {
        for (int j = 0; j < COL_A; j++)
        {
            *(A + i * COL_A + j) = (rand() / float(RAND_MAX)) * RANGE;
        }
    }
    for (int i = 0; i < ROW_B; i++)
    {
        for (int j = 0; j < COL_B; j++)
        {
            *(B + i * COL_B + j) = (rand() / float(RAND_MAX)) * RANGE;
        }
    }
}

//CPU串行版本矩阵乘法
void matMulCPU(float *A, float *B, float *C)
{
    for (int i = 0; i < ROW_A; i++)
    {
        for (int j = 0; j < COL_B; j++)
        {
            float sum = 0;
            for (int k = 0; k < COL_A; k++)
            {
                sum += A[i * COL_A + k] * B[k * COL_A + j];
            }
            C[i * ROW_A + j] = sum;
        }
    }
}

void displayMat(float* A, int row, int col)
{
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            cout << A[i * row + j] << " ";
        }
        cout << endl;
    }
}

//设备端函数
__global__ static void CUDAkernel(const float *a, const float *b, float *c)
{
    //block内的threadID
    const int tid = threadIdx.x;
    //blockID
    const int bid = blockIdx.x;
    //全局threadID
    const int idx = bid * THREAD_NUM + tid;
    const int row = idx / COL_B;
    const int column = idx % COL_B;
    //计算矩阵乘法
    if (row < ROW_A && column < COL_B)
    {
        float t = 0;
        for (int i = 0; i < COL_A; i++)
        {
            t += a[row * COL_A + i] * b[i * COL_B + column];
        }
        c[row * COL_B + column] = t;
    }
}

int main()
{
    //定义矩阵
    float *a, *b, *c, *d;
    //分配主机端内存
    a = (float *)malloc(sizeof(float) * ROW_A * COL_A);
    b = (float *)malloc(sizeof(float) * ROW_B * COL_B);
    c = (float *)malloc(sizeof(float) * ROW_A * COL_B);
    d = (float *)malloc(sizeof(float) * ROW_A * COL_B);
    float *cuda_a, *cuda_b, *cuda_c;
    //分配设备端显存
    hipMalloc((void **)&cuda_a, sizeof(float) * ROW_A * COL_A);
    hipMalloc((void **)&cuda_b, sizeof(float) * ROW_B * COL_B);
    hipMalloc((void **)&cuda_c, sizeof(float) * ROW_A * COL_B);
    //生成矩阵 a, b
    generateMatrix(a, b);
    cout << "Matrix A:" << endl;
    displayMat(a, ROW_A, COL_A);
    cout << "Matrix B:" << endl;
    displayMat(b, ROW_B, COL_B);
    //开始计算并行时间
    auto par_start = system_clock::now();
    //cudaMemcpyHostToDevice 从内存复制到显存
    //cudaMemcpyDeviceToHost 从显存复制到内存
    hipMemcpy(cuda_a, a, sizeof(float) * ROW_A * COL_A, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(float) * ROW_B * COL_B, hipMemcpyHostToDevice);
    //设备端函数
    CUDAkernel<<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_c);
    //cudaMemcpy 将结果从显存中复制回内存
    //比较加速比
    hipMemcpy(c, cuda_c, sizeof(float) * ROW_A * COL_B, hipMemcpyDeviceToHost);
    auto par_end = system_clock::now();

    auto cpu_start = system_clock::now();
    matMulCPU(a, b, d);
    auto cpu_end = system_clock::now();
    
    auto par_time = duration_cast<nanoseconds>(par_end - par_start);
    auto cpu_time = duration_cast<nanoseconds>(cpu_end - cpu_start);
    cout << "Matrix C:" << endl;
    displayMat(c, ROW_A, COL_B);
    cout << "Matrix D:" << endl;
    displayMat(d, ROW_A, COL_B);
    cout << "Parallel time: " << par_time.count() << endl;
    cout << "Cpu time: " << cpu_time.count() << endl;
    cout << "Speedup: " << cpu_time.count() /par_time.count() << endl;
    //Free
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);
}
